#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <vector>
#include <queue>

using namespace std;

#include "swsharp/swsharp.h"
#include "swsharp/cuda_utils.h"
#include "table_node.h"
#include "ac_table.h"
#include "ac_cuda_utils.h"

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer);
// ***************************************************************************

// ***************************************************************************
// PRIVATE
typedef vector<vector<int> > Candidates;
typedef vector<int> Candidate;

// TODO: to void
static TableGpu* copyTableToGpu(TabNode* table, TableGpu** hostCopy);
static void deleteTableGpu(TableGpu* table, TableGpu* hostCopy);

static void chainGpuCreate(Chain* chain, ChainGpu** chainD, ChainGpu** chainH);
static void chainGpuDelete(ChainGpu* chainD, ChainGpu* chainH);

__global__ static void findCandidates(TableGpu** automata, 
    int automataLen, ChainGpu** database, int databaseLen,
    int* candidates);

__global__ static void findCandidatesBlocks(TableGpu** automata, 
    int automatonIndex, ChainGpu** database, int databaseLen,
    int numBlocks, int* candidates);

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer) {

    fprintf(stderr,"Creating indices\n");
    vector<TabNode*>* aut = static_cast<vector<TabNode*>*>(automata);

    vector<TableGpu*> gpuTables;
    vector<TableGpu*> hostTables;

    gpuTables.reserve(automataLen);
    hostTables.reserve(automataLen);

    vector<ChainGpu*> gpuChains;
    vector<ChainGpu*> hostChains;

    int numTargets = databaseLen - databaseStart;

    gpuChains.reserve(numTargets);
    hostChains.reserve(numTargets);

    //**************************************************************************
    // SEND AUTOMATA TO GPU
    fprintf(stderr,"Sending automata to gpu\n");
    for (int i = 0; i < automataLen; ++i) {
        TabNode* autH = (*aut)[i];

        TableGpu* hostCopy;

        gpuTables.push_back(copyTableToGpu(autH, &hostCopy));
        hostTables.push_back(hostCopy);
        // fprintf(stderr,"%p\n", gpuTables[i]);
    }
    fprintf(stderr,"Done\n");

    TableGpu** gpuTablesD;
    CUDA_SAFE_CALL(hipMalloc(&gpuTablesD, sizeof(TableGpu*) * automataLen));
    CUDA_SAFE_CALL(hipMemcpy(gpuTablesD, &gpuTables[0], 
        sizeof(TableGpu*) * automataLen, TO_GPU));    

    //**************************************************************************

    //**************************************************************************
    // SEND DATABASE TO GPU

    fprintf(stderr,"Sending database to gpu.\n");
    for (int i = databaseStart; i < databaseLen; ++i) {
        ChainGpu* chainD;
        ChainGpu* chainH;

        chainGpuCreate(database[i], &chainD, &chainH);

        gpuChains.push_back(chainD);
        hostChains.push_back(chainH);
    }
    fprintf(stderr,"DOne\n");

    ChainGpu** chainsGpuD;
    CUDA_SAFE_CALL(hipMalloc(&chainsGpuD, sizeof(ChainGpu*) * numTargets));
    CUDA_SAFE_CALL(hipMemcpy(chainsGpuD, &gpuChains[0], 
        sizeof(ChainGpu*) * numTargets, TO_GPU));

    //**************************************************************************
    fprintf(stderr,"Allocating candidates.\n");
    int* candidatesD;
    CUDA_SAFE_CALL(hipMalloc(&candidatesD, sizeof(int) * 5001 * automataLen));
    int* candidatesH = (int*) malloc(sizeof(int) * 5001 * automataLen);

    //**************************************************************************
    // INVOKE KERNEL
    int grid_x = min(5000, databaseLen);
    dim3 dimGrid(grid_x, 1, 1);
    dim3 dimBlock(1, 1, 1);

    for (int i = 0; i < automataLen; ++i) {
        fprintf(stderr,"Invoking kernel\n");
        
        findCandidatesBlocks<<<dimGrid, dimBlock>>>(gpuTablesD, i, chainsGpuD,
            databaseLen, grid_x, candidatesD);
    }

    // int grid_x = automataLen;
    // int block_x = 1;

    // dim3 dimGrid(grid_x,1,1);
    // // fprintf(stderr,"Automata len: %d\n", automataLen);
    // dim3 dimBlock(block_x,1,1);
    // findCandidates<<<dimGrid, dimBlock>>>(gpuTablesD, automataLen, chainsGpuD, 
    //     numTargets, candidatesD);

    CUDA_SAFE_CALL(hipMemcpy(candidatesH, candidatesD, 
        sizeof(int) * 5001 * automataLen, FROM_GPU));

    //**************************************************************************
    // EXTRACT CANDIDATES

    fprintf(stderr,"Extracting candidates\n");
    Candidates* candidates = new Candidates();
    candidates->reserve(automataLen);

    for (int i = 0; i < automataLen; ++i) {
        Candidate queryCandidates;

        for (int j = 0; j < grid_x; ++j) {
            if (candidatesH[i * 5001 + j + 1] > -1) {
                queryCandidates.push_back(candidatesH[i * 5001 +j + 1]);
            }
        }

        candidates->push_back(queryCandidates);
    }

    // for (int i = 0; i < automataLen; ++i) {
    //     Candidate queryCandidates;

    //     int size = candidatesH[i * 5001];
    //     // fprintf(stderr,"Size: %d\n", size);
    //     for (int j = 0; j < size; ++j) {
    //         // fprintf(stderr,"Kandidat: %d\n", candidatesH[i * 5001 + j + 1]);
    //         queryCandidates.push_back(candidatesH[i * 5001 + j + 1]);
    //         // fprintf(stderr, "Candidate is: %d\n", candidatesH[i * 5001 + j + 1]);
    //     }

    //     candidates->push_back(queryCandidates);
    // }

    free(candidatesH);
    fprintf(stderr,"Done\n");

    //**************************************************************************
    // CLEAN UP
    for (int i = 0; i < automataLen; ++i) {
        deleteTableGpu(gpuTables[i], hostTables[i]);
    }

    for (int i = 0; i < numTargets; ++i) {
        chainGpuDelete(gpuChains[i], hostChains[i]);
    }

    CUDA_SAFE_CALL(hipFree(gpuTablesD));
    CUDA_SAFE_CALL(hipFree(chainsGpuD));
    CUDA_SAFE_CALL(hipFree(candidatesD));

    gpuTables.clear();
    //**************************************************************************
    fprintf(stderr,"Done cleaning\n");
    return static_cast<void*>(candidates);
}
// ***************************************************************************

// ***************************************************************************
// PRIVATE
static TableGpu* copyTableToGpu(TabNode* table, TableGpu** hostCopy) {
    *hostCopy = (TableGpu*) malloc(sizeof(TableGpu));

    (*hostCopy)->numStates = table->numStates;
    (*hostCopy)->table = table->table;

    // flatten and copy positions vector
    int start = 0;
    vector<int> positions;
    vector<vector<uint16> > &v = table->positions;

    for (int i = 0; i < (*hostCopy)->numStates; ++i) {
        (*hostCopy)->table[i * TABLE_WIDTH + POSITIONS_START] = start;

        positions.insert(positions.end(), v[i].begin(), v[i].end());

        start += v[i].size();
    }

    uint16* positionsD;
    CUDA_SAFE_CALL(hipMalloc(&positionsD, sizeof(uint16) * positions.size()));

    CUDA_SAFE_CALL(hipMemcpy(positionsD, &positions[0], 
        sizeof(uint16) * positions.size(),
        hipMemcpyHostToDevice));

    (*hostCopy)->positions = positionsD;

    // state table
    int* statesD;
    CUDA_SAFE_CALL(hipMalloc(&statesD, 
        sizeof(int) * (*hostCopy)->numStates * TABLE_WIDTH));

    CUDA_SAFE_CALL(hipMemcpy(statesD, (*hostCopy)->table,
        sizeof(int) * (*hostCopy)->numStates * TABLE_WIDTH, 
        hipMemcpyHostToDevice));

    (*hostCopy)->table = statesD;

    TableGpu* autD;
    CUDA_SAFE_CALL(hipMalloc(&autD, sizeof(TableGpu)));

    CUDA_SAFE_CALL(hipMemcpy(autD, (*hostCopy), sizeof(TableGpu), 
        hipMemcpyHostToDevice));

    return autD;
}

static void deleteTableGpu(TableGpu* table, TableGpu* hostCopy) {
    CUDA_SAFE_CALL(hipFree(hostCopy->table));
    CUDA_SAFE_CALL(hipFree(hostCopy->positions));
    CUDA_SAFE_CALL(hipFree(table));    

    free(hostCopy);
}

static void chainGpuCreate(Chain* chain, ChainGpu** chainD, ChainGpu** chainH) {
    (*chainH) = (ChainGpu*) malloc(sizeof(ChainGpu));

    int len = chainGetLength(chain);
    const char* codes = chainGetCodes(chain);

    int rest = len % 4;
    int mallocLen = rest > 0 ? 1 : 0;
    char4* codesH = (char4*) malloc(sizeof(char4) * (len / 4 + mallocLen));

    for (int i = 0; i < (len - rest) / 4; ++i) {
        codesH[i].x = codes[i * 4];
        codesH[i].y = codes[i * 4 + 1];
        codesH[i].z = codes[i * 4 + 2];
        codesH[i].w = codes[i * 4 + 3];
    }

    if (rest) {
        codesH[len / 4].x = codes[len-rest];

        if (rest >= 2) {
            codesH[len / 4].y = codes[len-rest + 1];
        } else {
            codesH[len / 4].y = 0;
        }

        if (rest == 3) {
            codesH[len / 4].z = codes[len-rest + 2];
        } else {
            codesH[len / 4].z = 0;
        }

        codesH[len / 4].w = 0;
    }

    char4* codesD;

    CUDA_SAFE_CALL(hipMalloc(&codesD, sizeof(char4) * (len / 4 + mallocLen)));
    CUDA_SAFE_CALL(hipMemcpy(codesD, codesH, 
        sizeof(char4) * (len / 4 + mallocLen), TO_GPU));

    (*chainH)->len = len;
    (*chainH)->codes = codesD;

    CUDA_SAFE_CALL(hipMalloc(chainD, sizeof(ChainGpu)));
    CUDA_SAFE_CALL(hipMemcpy(*chainD, *chainH, sizeof(ChainGpu), TO_GPU));
}

static void chainGpuDelete(ChainGpu* chainD, ChainGpu* chainH) {
    hipFree(chainH->codes);
    hipFree(chainD);

    free(chainH);
}
// ***************************************************************************

// ***************************************************************************
// GPU Modules

__global__ static void findCandidates(TableGpu** automata, 
    int automataLen, ChainGpu** database, int databaseLen,
    int* candidates) {

    int index = blockIdx.x;
    int candidatesSize = 0;

    if (index < automataLen) {
        candidates[index * 5001] = 0;
        int* table = automata[index]->table;

        for (int i = 0; i < databaseLen; ++i) {
            // printf("there is a db\n");
            int targetLen = database[i]->len;
            char4* codes = database[i]->codes;

            // if (index == 0 && threadIdx.x == 0) {
                // printf("Analyzing target %d\n", i);
                // printf("Target len %d\n", targetLen);
            // }

            // copy codes to shared memory
            // char4 ch4;
            // for (int k = threadIdx.x; k < targetLen / 4; k += 32) {
            //     ch4 = codes[k];

            //     // if (index == 0) {
            //         // printf("Thread %d Current chars: %d\n", threadIdx.x, k);
            //     // }

            //     int j = k * 4;

            //     // if (index == 0) {
            //     //     printf("Setting codesShared to %d\n", j);
            //     // }

            //     codesShared[j] = ch4.x;
            //     codesShared[j+1] = ch4.y;
            //     codesShared[j+2] = ch4.z;
            //     codesShared[j+3] = ch4.w;
            // }

            // // read the rest
            // if ((targetLen % 4) && threadIdx.x == 0) {
            //     int k = targetLen % 4;

            //     codesShared[targetLen - k] = codes[targetLen / 4].x;
            //     codesShared[targetLen - k + 1] = codes[targetLen / 4].y;
            //     codesShared[targetLen - k + 2] = codes[targetLen / 4].z;
            //     codesShared[targetLen - k + 3] = codes[targetLen / 4].w;
            // } else {

            // }

            // __syncthreads();

            // do transitions on the automaton and fill the candidates
            // if (threadIdx.x == 0) {
            int state = 0;
            int numHits = 0;
            char buff[4];
            // printf("targetlen %d\n", targetLen);
            int limit = targetLen % 4 ? targetLen / 4 + 1 : targetLen / 4;
            // printf("limit %d\n", limit);

            for (int k = 0; k < limit; ++k) {
                char4 in = codes[k];
                buff[0] = in.x;
                buff[1] = in.y;
                buff[2] = in.z;
                buff[3] = in.w;
                // printf("Current code %d\n", code);
                for (int j = 0; j < 4; ++j) {
                    char code = buff[j];

                    while(table[state * TABLE_WIDTH + code] == 0 && state != 0) {
                        state = table[state * TABLE_WIDTH + FAIL_COL];
                    }

                    if (state == table[state * TABLE_WIDTH + code]) {
                        continue;
                    }

                    state = table[state * TABLE_WIDTH + code];
                    if (table[state * TABLE_WIDTH + FINAL_COL]) {
                        numHits++;
                    }
                }

            }

            if (numHits > 0) {
                candidates[index * 5001 + (candidatesSize % 5000 + 1)] = i;
                candidatesSize = min(candidatesSize + 1, 5000);
                // candidatesSize = candidatesSize + 1 > 5000 ? 5000 : candidatesSize + 1;
            }
                // printf("Thread id: %d num hits: %d\n", index, numHits);
            // }
            // __syncthreads();
        }

        // printf("Candidates size: %d\n", candidatesSize);
        if (threadIdx.x == 0)
            candidates[index * 5001] = candidatesSize;
    }
}


__global__ static void findCandidatesBlocks(TableGpu** automata, 
    int automataIndex, ChainGpu** database, int databaseLen,
    int numBlocks, int* candidates) {

    int index = automataIndex;

    candidates[index * 5001] = 0;
    candidates[index * 5001 + 1 + blockIdx.x] = -1;
    int* table = automata[index]->table;

    for (int i = blockIdx.x; i < databaseLen; i += numBlocks) {
        // printf("there is a db\n");
        int targetLen = database[i]->len;
        char4* codes = database[i]->codes;

        // do transitions on the automaton and fill the candidates
        int state = 0;
        int numHits = 0;
        char buff[4];
        // printf("targetlen %d\n", targetLen);
        int limit = targetLen % 4 ? targetLen / 4 + 1 : targetLen / 4;
        // printf("limit %d\n", limit);

        for (int k = 0; k < limit; ++k) {
            char4 in = codes[k];
            buff[0] = in.x;
            buff[1] = in.y;
            buff[2] = in.z;
            buff[3] = in.w;
            // printf("Current code %d\n", code);
            for (int j = 0; j < 4; ++j) {
                char code = buff[j];

                while(table[state * TABLE_WIDTH + code] == 0 && state != 0) {
                    state = table[state * TABLE_WIDTH + FAIL_COL];
                }

                if (state == table[state * TABLE_WIDTH + code]) {
                    continue;
                }

                state = table[state * TABLE_WIDTH + code];
                if (table[state * TABLE_WIDTH + FINAL_COL]) {
                    numHits++;
                }
            }

        }

        if (numHits > 0) {
            candidates[index * 5001 + 1 + blockIdx.x] = i;
        }
    }
}

// ***************************************************************************
