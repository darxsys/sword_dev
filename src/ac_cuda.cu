#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <vector>
#include <queue>

using namespace std;

#include "swsharp/swsharp.h"
#include "swsharp/cuda_utils.h"
#include "table_node.h"
#include "ac_table.h"
#include "ac_cuda_utils.h"

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer);
// ***************************************************************************

// ***************************************************************************
// PRIVATE
typedef vector<vector<int> > Candidates;
typedef vector<int> Candidate;

// TODO: to void
static TableGpu* copyTableToGpu(TabNode* table, TableGpu** hostCopy);
static void deleteTableGpu(TableGpu* table, TableGpu* hostCopy);

static void chainGpuCreate(Chain* chain, ChainGpu** chainD, ChainGpu** chainH);
static void chainGpuDelete(ChainGpu* chainD, ChainGpu* chainH);

__global__ static void findCandidates(TableGpu** automata, 
    int automataLen, ChainGpu** database, int databaseLen,
    int* candidates);

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer) {

    vector<TabNode*>* aut = static_cast<vector<TabNode*>*>(automata);

    vector<TableGpu*> gpuTables;
    vector<TableGpu*> hostTables;

    gpuTables.reserve(automataLen);
    hostTables.reserve(automataLen);

    vector<ChainGpu*> gpuChains;
    vector<ChainGpu*> hostChains;

    int numTargets = databaseLen - databaseStart;

    gpuChains.reserve(numTargets);
    hostChains.reserve(numTargets);

    //**************************************************************************
    // SEND AUTOMATA TO GPU
    
    for (int i = 0; i < automataLen; ++i) {
        TabNode* autH = (*aut)[i];

        TableGpu* hostCopy;

        gpuTables.push_back(copyTableToGpu(autH, &hostCopy));
        hostTables.push_back(hostCopy);
        // printf("%p\n", gpuTables[i]);
    }

    TableGpu** gpuTablesD;
    CUDA_SAFE_CALL(hipMalloc(&gpuTablesD, sizeof(TableGpu*) * automataLen));
    CUDA_SAFE_CALL(hipMemcpy(gpuTablesD, &gpuTables[0], 
        sizeof(TableGpu*) * automataLen, TO_GPU));    

    //**************************************************************************

    //**************************************************************************
    // SEND DATABASE TO GPU

    for (int i = databaseStart; i < databaseLen; ++i) {
        ChainGpu* chainD;
        ChainGpu* chainH;

        chainGpuCreate(database[i], &chainD, &chainH);

        gpuChains.push_back(chainD);
        hostChains.push_back(chainH);
    }

    ChainGpu** chainsGpuD;
    CUDA_SAFE_CALL(hipMalloc(&chainsGpuD, sizeof(ChainGpu*) * numTargets));
    CUDA_SAFE_CALL(hipMemcpy(chainsGpuD, &gpuChains[0], 
        sizeof(ChainGpu*) * numTargets, TO_GPU));

    //**************************************************************************
    int* candidatesD;
    CUDA_SAFE_CALL(hipMalloc(&candidatesD, sizeof(int) * 5001 * automataLen));
    int* candidatesH = (int*) malloc(sizeof(int) * 5001 * automataLen);


    dim3 dimGrid(1,1,1);
    // TODO: change
    dim3 dimBlock(automataLen,1,1);

    findCandidates<<<dimGrid, dimBlock>>>(gpuTablesD, automataLen, chainsGpuD, 
        numTargets, candidatesD);

    CUDA_SAFE_CALL(hipMemcpy(candidatesH, candidatesD, 
        sizeof(int) * 5001 * automataLen, FROM_GPU));

    //**************************************************************************
    // EXTRACT CANDIDATES

    Candidates* candidates = new Candidates();
    candidates->reserve(automataLen);
    for (int i = 0; i < automataLen; ++i) {
        Candidate queryCandidates;

        int size = candidatesH[i * 5001];
        // printf("Size: %d\n", size);
        for (int j = 0; j < size; ++j) {
            // printf("Kandidat: %d\n", candidatesH[i * 5001 + j + 1]);
            queryCandidates.push_back(candidatesH[i * 5001 + j + 1]);
        }

        candidates->push_back(queryCandidates);
    }

    free(candidatesH);

    //**************************************************************************
    // CLEAN UP
    for (int i = 0; i < automataLen; ++i) {
        deleteTableGpu(gpuTables[i], hostTables[i]);
    }

    for (int i = 0; i < numTargets; ++i) {
        chainGpuDelete(gpuChains[i], hostChains[i]);
    }

    CUDA_SAFE_CALL(hipFree(gpuTablesD));
    CUDA_SAFE_CALL(hipFree(chainsGpuD));
    CUDA_SAFE_CALL(hipFree(candidatesD));

    gpuTables.clear();
    //**************************************************************************

    return static_cast<void*>(candidates);
}
// ***************************************************************************

// ***************************************************************************
// PRIVATE
static TableGpu* copyTableToGpu(TabNode* table, TableGpu** hostCopy) {
    *hostCopy = (TableGpu*) malloc(sizeof(TableGpu));

    (*hostCopy)->numStates = table->numStates;
    (*hostCopy)->table = table->table;

    // flatten and copy positions vector
    int start = 0;
    vector<int> positions;
    vector<vector<uint16> > &v = table->positions;

    for (int i = 0; i < (*hostCopy)->numStates; ++i) {
        (*hostCopy)->table[i * TABLE_WIDTH + POSITIONS_START] = start;

        positions.insert(positions.end(), v[i].begin(), v[i].end());

        start += v[i].size();
    }

    uint16* positionsD;
    CUDA_SAFE_CALL(hipMalloc(&positionsD, sizeof(uint16) * positions.size()));

    CUDA_SAFE_CALL(hipMemcpy(positionsD, &positions[0], 
        sizeof(uint16) * positions.size(),
        hipMemcpyHostToDevice));

    (*hostCopy)->positions = positionsD;

    // state table
    int* statesD;
    CUDA_SAFE_CALL(hipMalloc(&statesD, sizeof(int) * (*hostCopy)->numStates * TABLE_WIDTH));

    CUDA_SAFE_CALL(hipMemcpy(statesD, (*hostCopy)->table,
        sizeof(int) * (*hostCopy)->numStates * TABLE_WIDTH, 
        hipMemcpyHostToDevice));

    (*hostCopy)->table = statesD;

    TableGpu* autD;
    CUDA_SAFE_CALL(hipMalloc(&autD, sizeof(TableGpu)));

    CUDA_SAFE_CALL(hipMemcpy(autD, (*hostCopy), sizeof(TableGpu), 
        hipMemcpyHostToDevice));

    // printf("autd %p\n", autD);
    return autD;
}

static void deleteTableGpu(TableGpu* table, TableGpu* hostCopy) {
    CUDA_SAFE_CALL(hipFree(hostCopy->table));
    CUDA_SAFE_CALL(hipFree(hostCopy->positions));
    CUDA_SAFE_CALL(hipFree(table));    

    free(hostCopy);
}

static void chainGpuCreate(Chain* chain, ChainGpu** chainD, ChainGpu** chainH) {
    (*chainH) = (ChainGpu*) malloc(sizeof(ChainGpu));

    int len = chainGetLength(chain);
    const char* codesH = chainGetCodes(chain);
    char* codesD;

    CUDA_SAFE_CALL(hipMalloc(&codesD, sizeof(char) * len));
    CUDA_SAFE_CALL(hipMemcpy(codesD, codesH, sizeof(char) * len, TO_GPU));

    (*chainH)->len = len;
    (*chainH)->codes = codesD;

    CUDA_SAFE_CALL(hipMalloc(chainD, sizeof(ChainGpu)));
    CUDA_SAFE_CALL(hipMemcpy(*chainD, *chainH, sizeof(ChainGpu), TO_GPU));
}

static void chainGpuDelete(ChainGpu* chainD, ChainGpu* chainH) {
    hipFree(chainH->codes);
    hipFree(chainD);

    free(chainH);
}
// ***************************************************************************

// ***************************************************************************
// GPU Modules

__global__ static void findCandidates(TableGpu** automata, 
    int automataLen, ChainGpu** database, int databaseLen,
    int* candidates) {

    int index = threadIdx.x;

    int candidatesSize = 0;

    if (index < automataLen) {
        // printf("Inside\n");
        candidates[index * 5001] = 0;

        int* table = automata[index]->table;

        for (int i = 0; i < databaseLen; ++i) {
            // printf("there is a db\n");
            int targetLen = database[i]->len;
            char* codes = database[i]->codes;

            int state = 0;
            int numHits = 0;
            // do transitions on the automaton and fill candidates

            // printf("targetlen %d\n", targetLen);
            for (int k = 0; k < targetLen; ++k) {
                int code = codes[k];

                while(table[state * TABLE_WIDTH + code] == 0 && state != 0) {
                    // fprintf(stderr, "no transition\n");
                    state = table[state * TABLE_WIDTH + FAIL_COL];
                }

                if (state == table[state * TABLE_WIDTH + code]) {
                    // printf("Character %c continuing\n", c + 'A');
                    continue;
                }

                state = table[state * TABLE_WIDTH + code];
                // fprintf(stderr, "Current state: %d\n", state);
                if (table[state * TABLE_WIDTH + FINAL_COL]) {
                    // fprintf(stderr, "A hit. Sign: %c\n", c + 'A');
                    // printf("A hit\n");
                    numHits++;
                }

            }

            if (numHits > 0) {
                candidates[index * 5001 + (candidatesSize + 1) % 5000] = i;
                candidatesSize++;
                candidates[index * 5001] = candidatesSize;
            }
        }

    }

    // printf("heklo worls\n");
    return;
}

// ***************************************************************************
