#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <vector>
#include <queue>

using namespace std;

#include "swsharp/swsharp.h"
#include "table_node.h"
#include "ac_table.h"
#include "ac_cuda_utils.h"

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer);
// ***************************************************************************

// ***************************************************************************
// PRIVATE
static TableGpu* copyTableToGpu(TabNode* table);
static void deleteTableGpu(TableGpu* table);

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer) {

    vector<TabNode*>* aut = static_cast<vector<TabNode*>*>(automata);

    for (int i = 0; i < automataLen; ++i) {
        TabNode* autH = (*aut)[i];
        TableGpu* tab = copyTableToGpu(autH);

        deleteTableGpu(tab);
    }

    return NULL;
}
// ***************************************************************************

// ***************************************************************************
// PRIVATE

static TableGpu* copyTableToGpu(TabNode* table) {
    TableGpu* copyAut = (TableGpu*) malloc(sizeof(TableGpu));

    copyAut->numStates = table->numStates;
    copyAut->table = table->table;

    int* states;
    hipMalloc(&states, sizeof(int) * copyAut->numStates);
    hipMemcpy(states, copyAut->table, sizeof(int) * copyAut->numStates,
        hipMemcpyHostToDevice);

    copyAut->table = states;

    TableGpu* autD;
    hipMalloc(&autD, sizeof(TableGpu));
    hipMemcpy(autD, copyAut, sizeof(TableGpu), 
        hipMemcpyHostToDevice);
    
    return autD;
}

static void deleteTableGpu(TableGpu* table) {
    hipFree(table->table);
    hipFree(table);    
}

// ***************************************************************************
