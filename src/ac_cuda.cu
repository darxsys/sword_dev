#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <iostream>
#include <vector>
#include <queue>

using namespace std;

#include "swsharp/swsharp.h"
#include "table_node.h"
#include "ac_table.h"
#include "ac_cuda_utils.h"

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer);
// ***************************************************************************

// ***************************************************************************
// PRIVATE
static TableGpu* copyTableToGpu(TabNode* table);
static void deleteTableGpu(TableGpu* table);

__global__ static void findCandidates(TableGpu* automata, int automataLen);

// ***************************************************************************
// PUBLIC
extern void* indicesTableCreateGpu(Chain** database, 
    int databaseStart, int databaseLen, void* automata,
    int automataLen, int seedLen, Scorer* scorer) {

    vector<TabNode*>* aut = static_cast<vector<TabNode*>*>(automata);
    vector<TableGpu*> gpuTables;
    gpuTables.reserve(automataLen);

    for (int i = 0; i < automataLen; ++i) {
        TabNode* autH = (*aut)[i];
        gpuTables.push_back(copyTableToGpu(autH));

    }

    dim3 dimGrid(1,1,1);
    dim3 dimBlock(1,1,1);

    TableGpu* gpuTablesD;
    hipMalloc(&gpuTablesD, sizeof(TableGpu*) * automataLen);
    hipMemcpy(gpuTablesD, &gpuTables[0], 
        sizeof(TableGpu*) * automataLen, 
        hipMemcpyHostToDevice);

    findCandidates<<<dimGrid, dimBlock>>>(gpuTablesD, automataLen);

    // clean up
    hipFree(gpuTablesD);
    for (int i = 0; i < automataLen; ++i) {
        deleteTableGpu(gpuTables[i]);
    }
    gpuTables.clear();
    return NULL;
}
// ***************************************************************************

// ***************************************************************************
// PRIVATE

static TableGpu* copyTableToGpu(TabNode* table) {
    TableGpu* copyAut = (TableGpu*) malloc(sizeof(TableGpu));

    copyAut->numStates = table->numStates;
    copyAut->table = table->table;

    int* states;
    hipMalloc(&states, sizeof(int) * copyAut->numStates);
    hipMemcpy(states, copyAut->table, sizeof(int) * copyAut->numStates,
        hipMemcpyHostToDevice);

    copyAut->table = states;

    // flatten and copy positions vector
    int start = 0;
    vector<int> positions;
    vector<vector<uint16> > &v = table->positions;

    for (int i = 0; i < copyAut->numStates; ++i) {
        copyAut->table[i * TABLE_WIDTH + POSITIONS_START] = start;

        positions.insert(positions.end(), v[i].begin(), v[i].end());

        start += v[i].size();
    }

    uint16* positionsD;
    hipMalloc(&positionsD, sizeof(uint16) * positions.size());
    hipMemcpy(positionsD, &positions[0], 
        sizeof(uint16) * positions.size(),
        hipMemcpyHostToDevice);

    copyAut->positions = positionsD;

    TableGpu* autD;
    hipMalloc(&autD, sizeof(TableGpu));
    hipMemcpy(autD, copyAut, sizeof(TableGpu), 
        hipMemcpyHostToDevice);

    delete copyAut;
    return autD;
}

static void deleteTableGpu(TableGpu* table) {
    hipFree(table->table);
    hipFree(table->positions);
    hipFree(table);    
}

// ***************************************************************************

// ***************************************************************************
// GPU Modules

__global__ static void findCandidates(TableGpu* automata, int automataLen) {
    return;
}

// ***************************************************************************
